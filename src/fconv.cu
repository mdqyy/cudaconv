#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2012, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */
#include "fconv.hpp"
#include <hip/hip_runtime_api.h>
#include <cutil.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <opencv2/core/core.hpp>
#include <opencv2/gpu/gpu.hpp>


/* 
 * fconv
 * Is a specialised convolution routine for computing the responses to a 
 * set of features provided by the PartsBasedDetector
 */
 template <const unsigned int block_size>
__global__ void fconv_kernel(float **features, float **filters, float **responses, const unsigned int rwidth, const unsigned int rheight, const unsigned int feature_width, const unsigned int filter_width, const unsigned int filter_height) {

    // shared memory for the filter
    __shared__ float sfilter[2*block_size];
    // shared memory for the output of the multiplication
    __shared__ float sresponse[block_size];

    // get the thread indices into the filter
    const int idx = threadIdx.y*blockDim.x + threadIdx.x;

    // dereference this block's filter and feature
    const float *filter   = filters[blockIdx.x];
    const float *feature  = features[blockIdx.y];
    float *response = responses[blockIdx.y*gridDim.x + blockIdx.x]; 

    // initialise the response array
    sresponse[idx] = 0;

    // copy the filter into shared memory
    sfilter[idx]  = filter[idx];
    if (threadIdx.y+blockDim.y < filter_height) sfilter[idx+block_size] = filter[idx+block_size];
    __syncthreads();

    // loop over all heights
    for (int i = 0; i < rheight; ++i) {

        // loop over all widths
        for (int j = 0; j < rwidth; ++j) {
            
            sresponse[idx] =  feature[(i+threadIdx.y)*feature_width + j*filter_width + threadIdx.x] * sfilter[idx];
            if (threadIdx.y+blockDim.y < filter_height) sresponse[idx] +=  feature[(i+threadIdx.y+blockDim.y)*feature_width + j*filter_width + threadIdx.x] * sfilter[idx+block_size];
            __syncthreads();

            // reduce the response to a single value
            if (block_size >= 512)  {if (idx < 256)  {sresponse[idx] = sresponse[idx+256];  } __syncthreads(); }
            if (block_size >= 256)  {if (idx < 128)  {sresponse[idx] = sresponse[idx+128];  } __syncthreads(); }
            if (block_size >= 128)  {if (idx < 64)   {sresponse[idx] = sresponse[idx+64];   } __syncthreads(); }
            
            // unwrap last warp execution path so we no longer need to wait for
            // the other threads
            if (idx < 32) {
                if (block_size >= 64) sresponse[idx] = sresponse[idx+32];
                if (block_size >= 32) sresponse[idx] = sresponse[idx+16];
                if (block_size >= 16) sresponse[idx] = sresponse[idx+8];
                if (block_size >= 8)  sresponse[idx] = sresponse[idx+4];
                if (block_size >= 4)  sresponse[idx] = sresponse[idx+2];
                if (block_size >= 2)  sresponse[idx] = sresponse[idx+1];
            }
            // write the final result out to global memory
            if (idx == 0) response[j*rwidth+i] = sresponse[0];
        }
    }
    return;
}


void fconv(const std::vector<cv::gpu::DevMem2D_<float> > dev_features, 
           const std::vector<cv::gpu::DevMem2D_<float> > dev_filters, 
           std::vector<std::vector<cv::gpu::DevMem2D_<float> > > dev_responses) {

    // strip the data from the device pointers
    int nfeatures = dev_features.size();
    int nfilters  = dev_filters.size();

    float **features_d;
    float **features_h = new float *[nfeatures];
    CUDA_SAFE_CALL(hipMalloc((void **)&features_d, sizeof(float *) * nfeatures));
    for (int i = 0; i < nfeatures; ++i) features_h[i] = dev_features[i].data;
    CUDA_SAFE_CALL(hipMemcpy((void *)features_d, (void *)features_h, sizeof(float *) * nfeatures, hipMemcpyHostToDevice));
    
    
    float **filters_d;
    float **filters_h  = new float *[nfilters];
    CUDA_SAFE_CALL(hipMalloc((void **)&filters_d, sizeof(float *) * nfilters));
    for (int j = 0; j < nfilters; ++j) filters_h[j] = dev_filters[j].data;
    CUDA_SAFE_CALL(hipMemcpy((void *)filters_d, (void *)filters_h, sizeof(float *) * nfilters, hipMemcpyHostToDevice));

    
    assert(dev_responses.size() == nfeatures);
    float **responses_d;
    float **responses_h = new float *[nfeatures*nfilters];
    CUDA_SAFE_CALL(hipMalloc((void **)&responses_d, sizeof(float *) * nfeatures*nfilters));
    for (int i = 0; i < nfeatures; ++i) {
        assert(dev_responses[i].size() == nfilters);
        for (int j = 0; j < nfilters; ++j) {
            responses_h[i*nfilters + j] = dev_responses[i][j].data;
        }
    }
    CUDA_SAFE_CALL(hipMemcpy((void *)responses_d, (void *)responses_h, sizeof(float *) * nfeatures*nfilters, hipMemcpyHostToDevice));
    
    // get the response height and width
    int rwidth  = dev_responses[0][0].cols;
    int rheight = dev_responses[0][0].rows; 

    // invoke the kernel
    dim3 grid(nfilters, nfeatures);
    //dim3 grid(1,1);
    dim3 threads(32*4, 4/2);
    //fconv_kernel<32, 32*4*4><<< grid, threads >>>(features, filters, responses, rwidth, rheight);
    fconv_kernel<32*4*4><<< grid, threads >>>(features_d, filters_d, responses_d, rwidth, rheight, 240*32, 32, 4);
    hipDeviceSynchronize();
    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // setdown
    /*
    free(features);
    free(filters);
    for (int i = 0; i < nfeatures; ++i) free(responses[i]);
    free(responses);
    */
}
